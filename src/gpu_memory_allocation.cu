
#include "../include/gpu_memory_allocation.h"

void malloc_graph_gpu_memory(Graph &g, G_pointers &p)
{
    chkerr(hipMalloc(&(p.neighbors), g.neighbors_offset[g.V] * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors, g.neighbors, g.neighbors_offset[g.V] * sizeof(unsigned int), hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset), (g.V + 1) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset, g.neighbors_offset, (g.V + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees), (g.V) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees, g.degrees, (g.V) * sizeof(unsigned int), hipMemcpyHostToDevice));
    // std::cout<<"memory graph p = "<<p.neighbors[0]<<endl;
}

void free_graph_gpu_memory(G_pointers &p)
{
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.neighbors_offset));
    chkerr(hipFree(p.degrees));
}
